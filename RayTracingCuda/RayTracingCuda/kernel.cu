#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <time.h>

#include "Vector3.h"
#include "ray.h"

#define STB_IMAGE_IMPLEMENTATION
#define STBI_MSC_SECURE_CRT
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ Vector3 ray_color(const ray& r) {
    Vector3 unit_direction = normalize(r.direction());
    float t = 0.5f * (unit_direction.y() + 1.0f); //force single precision with floats
    return (1.0f - t) * Vector3(1.0, 1.0, 1.0) = t * Vector3(0.5, 0.7, 1.0);
}

__global__ void render(Vector3* fb, int max_x, int max_y, Vector3 lower_left_corner, Vector3 horizontal, Vector3 vertical, Vector3 origin) {
    int i = threadIdx.x + blockIdx.x * blockDim.x; //Using the threadIdx and blockIdx CUDA built-in variables we identify the coordinates of each thread in the image (i,j)
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;

    float u = float(i) / float(max_x);
    float v = float(j) / float(max_y);
    ray r(origin, lower_left_corner +u * horizontal + v * vertical);

    fb[pixel_index] = ray_color(r);
}

int main() {
    int nx = 1200;
    int ny = 600;
    int tx = 8; //threadX
    int ty = 8; //threadY

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx * ny; //frame buffer - has a pixel count that is a multiple of 32 in order to fit into warps evenly.
    size_t fb_size = 3 * num_pixels * sizeof(Vector3); //framebuffer multiplied with 3 channels

    // allocate FB
    Vector3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);
    render <<<blocks, threads >>>(fb, nx, ny, Vector3(-2.0, -1.0, -1.0), Vector3(4.0,0.0,0.0), Vector3(0.0,2.0,0.0), Vector3(0.0,0.0,0.0));
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    uint8_t* pixels = new uint8_t[nx * ny * 3];

    int index = 0;

    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny - 1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * fb[pixel_index].x());
            int ig = int(255.99 * fb[pixel_index].y());
            int ib = int(255.99 * fb[pixel_index].z());
            //std::cout << ir << " " << ig << " " << ib << "\n";

            pixels[index++] = ir;
            pixels[index++] = ig;
            pixels[index++] = ib;

        }
    }

    stbi_write_png("cuda.png", nx, ny, 3, pixels, nx * 3);
    delete[] pixels;
    checkCudaErrors(hipFree(fb));
}