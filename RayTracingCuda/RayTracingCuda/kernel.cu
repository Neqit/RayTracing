#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>

#include <iostream>
#include <time.h>

#include "Vector3.h"
#include "ray.h"
#include "hittable_list.h"
#include "sphere.h"
#include "camera.h"

#define STB_IMAGE_IMPLEMENTATION
#define STBI_MSC_SECURE_CRT
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

#define RANDVEC3 Vector3(hiprand_uniform(local_rand_state),hiprand_uniform(local_rand_state),hiprand_uniform(local_rand_state))

__device__ Vector3 random_in_unit_sphere(hiprandState* local_rand_state) {
    Vector3 p;
    do {
        p = 2.0f * RANDVEC3 - Vector3(1, 1, 1);
    } while (p.length_squared() >= 1.0f);
    return p;
}


__device__ Vector3 ray_color(const ray& r, hittable** world, hiprandState *local_rand_state) {
    ray cur_ray = r;
    float cur_attenuation = 1.0f;
    //max depth = 50
    for (int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            Vector3 target = rec.p + rec.normal + random_in_unit_sphere(local_rand_state);
            cur_attenuation *= 0.5f;
            cur_ray = ray(rec.p, target - rec.p);
        }
        else {
            Vector3 unit_direction = normalize(cur_ray.direction());
            float t = 0.5f * (unit_direction.y() + 1.0f);
            Vector3 c = (1.0f - t) * Vector3(1.0, 1.0, 1.0) + t * Vector3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }

    return Vector3(0.0, 0.0, 0.0); //exceed depth
    
}

__global__ void render(Vector3* fb, int max_x, int max_y, int ns, camera** cam, hittable** world, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x; //Using the threadIdx and blockIdx CUDA built-in variables we identify the coordinates of each thread in the image (i,j)
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;

    hiprandState local_rand_state = rand_state[pixel_index];
    Vector3 col(0, 0, 0);
    for (int s = 0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v);
        col += ray_color(r, world, &local_rand_state);
    }

    rand_state[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = std::sqrt(col[0]);
    col[1] = std::sqrt(col[1]);
    col[2] = std::sqrt(col[2]);

    fb[pixel_index] = col;
}
//separate kernel for time measure
__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    //Each thread will receive same seed = same starting states
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void create_world(hittable** d_list, hittable** d_world, camera** d_camera) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_list) = new sphere(Vector3(0, 0, -1), 0.5);
        *(d_list + 1) = new sphere(Vector3(0, -100.5, -1), 100);
        *d_world = new hittable_list(d_list, 2);
        *d_camera = new camera();
    }
}


__global__ void free_world(hittable** d_list, hittable** d_world, camera** d_camera) {
    delete* (d_list);
    delete* (d_list + 1);
    delete* d_world;
    delete* d_camera;
}

int main() {
    int nx = 1200;
    int ny = 600;
    int ns = 100; //samples
    int tx = 8; //threadX
    int ty = 8; //threadY

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx * ny; //frame buffer - has a pixel count that is a multiple of 32 in order to fit into warps evenly.
    size_t fb_size = 3 * num_pixels * sizeof(Vector3); //framebuffer multiplied with 3 channels

    // allocate FB
    Vector3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    // allocate random state
    hiprandState* d_rand_state;
    checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));


    // make our world of hitables
    hittable** d_list;
    checkCudaErrors(hipMalloc((void**)&d_list, 2 * sizeof(hittable*)));
    hittable** d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hittable*)));
    camera** d_camera;
    checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));
    create_world<<<1, 1>>>(d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);
    render_init << <blocks, threads >> > (nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, nx, ny, ns, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    uint8_t* pixels = new uint8_t[nx * ny * 3];

    int index = 0;

    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny - 1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * fb[pixel_index].x());
            int ig = int(255.99 * fb[pixel_index].y());
            int ib = int(255.99 * fb[pixel_index].z());
            //std::cout << ir << " " << ig << " " << ib << "\n";

            pixels[index++] = ir;
            pixels[index++] = ig;
            pixels[index++] = ib;

        }
    }

    stbi_write_png("cuda_diffuse.png", nx, ny, 3, pixels, nx * 3);
    delete[] pixels;

    checkCudaErrors(hipDeviceSynchronize());
    free_world<<< 1, 1 >>>(d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(fb));

    hipDeviceReset();
}